/**
 * This program implements a parallel code via a kernel call using muti-streaming and atomic methods to do a matrix multiplication 
 * and prints out the execution time and the product of the two matrices with each element printed out to a 
 * file called “product.dat” in a tab-delimited, row/column format.
 * 
 * This is the original matrixMult program, and it would be compared with other programs using various optimization methods
 * later.
 *
 * Users are expected to enter three arguments: the executable file, the output file (which is product.dat), and
 * the width of the square matrics.
 *
 * @author Richard Zhang {zhank20@wfu.edu}
 * @date Apr.25, 2023
 * @assignment Lab 5
 * @course CSC 347
 **/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define NUM_STREAMS 4
#define TILE_WIDTH 4

__global__ void MatrixMulKernel(float* M, float* N, float* P, int Width);

int main(int argc, char *argv[]) {
    int width = 0;
    srand(time(NULL)); // seed the random number generator

    // Determine if there are three arguments on the command line
    if (argc < 3) {
        printf("Command line arguments are not enough: %s \n", argv[0]);
        return 1;
    }

    // Determine if the matrix width entered by users is legitimate
    if (atoi(argv[2]) <= 0) {
        printf("The matrix width should not be less than 1: %s \n", argv[2]);
        return 2;
    }

    // Initialize the three arrays: a and b are the input arrays, and c is the output array
    width = atoi(argv[2]);
    float *a = (float *)malloc(width * width * sizeof(float));
    float *b = (float *)malloc(width * width * sizeof(float));
    float *c = (float *)malloc(width * width * sizeof(float));

    float *dev_a, *dev_b, *dev_c;
    int size = width * width * sizeof(float);

    /* determine if the matrix width is larger than the tile width. If not, exit */
    if (width < TILE_WIDTH) {
        printf("The matrix width should be equal to or larger than the tile size, which is 16 in this case: %s \n", argv[2]);
        return 3;
    }

    // Initialize matrices a and b
    hipMalloc((void **)&dev_a, size);
    hipMalloc((void **)&dev_b, size);
    hipMalloc((void **)&dev_c, size);

    // Assign random float numbers to the two input arrays
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            // generate random numbers between 0 and 10.0
            a[i * width + j] = (float)rand() / RAND_MAX * 10.0;
            b[i * width + j] = (float)rand() / RAND_MAX * 10.0;
        }
    }

    // Create an array of CUDA streams
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipStreamCreate(&streams[i]);
    }

    // Calculate the size of submatrices
    int sub_width = width / NUM_STREAMS;
    int sub_size = sub_width * width * sizeof(float);

    /* determine the size of warm up grid and block */
    dim3 dimBlock_warmup(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid_warmup((width + dimBlock_warmup.x - 1) / dimBlock_warmup.x, (width + dimBlock_warmup.y - 1) / dimBlock_warmup.y);
    /* to warm up the GPU */
    MatrixMulKernel<<<dimGrid_warmup, dimBlock_warmup>>>(dev_a, dev_b, dev_c, width);
    hipDeviceSynchronize(); /* make sure the first kernel call has finished before starting the timer */

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start); /* start the timer */

    // Copy input matrices to GPU and perform matrix multiplication using multiple streams
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipMemcpyAsync(&dev_a[i * sub_width * width], &a[i * sub_width * width], sub_size, hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(&dev_b[i * sub_width], &b[i * sub_width], sub_size, hipMemcpyHostToDevice, streams[i]);

        // int block_size = 32;
        // dim3 dimBlock(block_size, block_size);
        // dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (width + dimBlock.y - 1) / dimBlock.y);
        /* determine the size of grid and block */
        dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
        dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (width + dimBlock.y - 1) / dimBlock.y);

        MatrixMulKernel<<<dimGrid, dimBlock, 0, streams[i]>>>(dev_a, dev_b, dev_c, width);
    }

    hipEventRecord(stop); /* end the timer */

    // Copy the result back to the host using multiple streams
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipMemcpyAsync(&c[i * sub_width * width], &dev_c[i * sub_width * width], sub_size, hipMemcpyDeviceToHost, streams[i]);
    }

    // Synchronize all streams
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipStreamSynchronize(streams[i]);
    }

    /* output the execution time of the kernel function to the terminal */
    float total_time = 0.0;
    hipEventElapsedTime(&total_time, start, stop);
    printf("Total execution time: %f seconds\n", total_time);
    
    // Move the output content to the output file, which is "product.dat"
    freopen(argv[1], "w", stdout);

    // Print the output array, which is the array c
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            printf("%f\t", c[i * width + j]);
        }
        printf("\n");
    }

    // Free memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    free(a);
    free(b);
    free(c);

    // Destroy CUDA streams
    for (int i = 0; i < NUM_STREAMS; ++i) {
        hipStreamDestroy(streams[i]);
    }

    return 0;
}

__global__ void MatrixMulKernel(float* M, float* N, float* P, int Width) {
    /* Initiate shared memory */
    __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];
    
    /* determine the indics of block and thread */
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    /* calculate row and col indics */
    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;
    
    /* loop over the M and N tiles to compute the P element */
    for (int p = 0; p < Width/TILE_WIDTH; ++p) {

        /* load M and N tiles into shared memory */
        ds_M[ty][tx] = M[row * Width + p * TILE_WIDTH + tx];
        ds_N[ty][tx] = N[(p * TILE_WIDTH + ty)* Width + col];
        __syncthreads();
        
        /* conduct the dot product between M and N tiles */
        for (int i = 0; i < TILE_WIDTH; ++i) {
            /* using attomicAdd() */
            float product = ds_M[ty][i] * ds_N[i][tx];
            atomicAdd(&P[row * Width + col], product);
        }
        
        /* wait for all threads in block to finish */
        __syncthreads();
    }
}
